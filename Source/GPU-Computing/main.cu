#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <chrono>
#include "data.h"
#include "kernel.h"
#include "sequential.h"

using namespace std;

int main(void) {

    //Kernel part
    hipSetDevice(0);

    int device;
    hipGetDevice(&device);
    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    printf("Max Threads per Block:%d\n", props.maxThreadsPerBlock);
    printf("Max Blocks per Multiprocessor:%d\n", props.maxBlocksPerMultiProcessor);
    printf("Max Shared Memory size per Block:%d\n", props.sharedMemPerBlock);

    int nodes = 512;
    int* matrix = (int*)malloc(nodes * nodes * sizeof(int*));
    for (int i = 0; i < nodes; i++) {
        matrix[i] = 9;
    }

    printf("Shared Memory size per Block:%d bytes\n", sizeof(int) * nodes + sizeof(int) * nodes * 2 + sizeof(bool) * nodes + sizeof(bool));
    generateRandomGraph(matrix, nodes);
    printf("Initial matrix loaded\n");

    /*for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            printf("%d", matrix[i * nodes + j]);
        }
        printf("\n");
    }*/

    /*int index = 0;
    for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            matrix[index] = adj_matrix[i][j];
            index++;
        }
    }*/

    int* gpu_matrix;
    hipError_t hipError_t = hipMalloc(&gpu_matrix, nodes * nodes * sizeof(int));

    if (hipError_t != hipSuccess) {
        printf("Errore during matrix allocation on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Matrix allocation on GPU completed\n");

    hipError_t = hipMemcpy(gpu_matrix, matrix, nodes * nodes * sizeof(int), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
        printf("Error during matrix copy on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Matrix copy on GPU completed\n");

    int* resultsMatrix;
    int* results = (int*)malloc(nodes * nodes * sizeof(int));
    hipError_t = hipMalloc(&resultsMatrix, nodes * nodes * sizeof(int));

    if (hipError_t != hipSuccess) {
        printf("Error during results matrix allocation on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Results matrix allocation completed\n");

    using clock = std::chrono::system_clock;
    using ms = std::chrono::duration<double, std::milli>;
    auto before = clock::now();

    //KERNEL V1 PART

    printf("\n\nKERNEL V1 PART\n\n");

    shortestPathsParallel <<<1, nodes >>> (gpu_matrix, nodes, resultsMatrix);
    hipError_t = hipGetLastError();

    if (hipError_t != hipSuccess) {
        printf("Error during kernel launch: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }

    hipError_t = hipDeviceSynchronize();
    if (hipError_t != hipSuccess) {
        printf("Kernel syncronization returned error: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }

    ms duration = clock::now() - before;

    hipError_t = hipMemcpy(results, resultsMatrix, nodes * nodes * sizeof(int), hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess) {
        printf("Error during results copy on Host: %s\n", hipGetErrorString(hipError_t));
    }
    printf("Results copy on Host completed\n");

    /*for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            printf("%d", results[i * nodes + j]);
        }
        printf("\n");
    }*/

    printf("Kernel execution time: %f milliseconds\n", duration.count());

    // KERNEL V2 PART

    printf("\n\nKERNEL V2 PART\n\n");

    before = clock::now();
    shortestPathsParallelV2 <<<nodes, nodes, sizeof(int) * nodes + sizeof(int) * nodes * 2 + sizeof(bool) * nodes + sizeof(bool) >> > (gpu_matrix, resultsMatrix);
    hipError_t = hipGetLastError();

    if (hipError_t != hipSuccess) {
        printf("Error during kernel V2 launch: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }

    hipError_t = hipPeekAtLastError();
    if (hipError_t != hipSuccess) {
        printf("Error during kernel V2 execution: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }

    hipError_t = hipDeviceSynchronize();
    if (hipError_t != hipSuccess) {
        printf("Kernel V2 syncronization returned error: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }

    duration = clock::now() - before;

    hipError_t = hipMemcpy(results, resultsMatrix, nodes * nodes * sizeof(int), hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess) {
        printf("Error during results copy on Host: %s\n", hipGetErrorString(hipError_t));
    }
    printf("Results copy on Host completed\n");

    /*for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            printf("%d", results[i * nodes + j]);
        }
        printf("\n");
    }*/

    hipFree(resultsMatrix);
    hipFree(gpu_matrix);
    printf("Kernel V2 execution time: %f milliseconds\n", duration.count());

    //SEQUENTIAL PART

    // Results matrix re-initialization
    for (int i = 0; i < nodes * nodes; i++) {
        results[i] = 0;
    }

    printf("\n\nSEQUENTIAL PART\n\n");

    before = clock::now();
    shortestPathsSequential(matrix, nodes, results);
    duration = clock::now() - before;

    /*printf("Sequential algorithm completed\n");
    for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            printf("%d", results[i * nodes + j]);
        }
        printf("\n");
    }*/

    printf("Sequential execution time: %f milliseconds\n", duration.count());
    free(results);
    free(matrix);
    return 0;
}