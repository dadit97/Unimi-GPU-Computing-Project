#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <chrono>
#include "data.h"
#include "kernel.h"
#include "sequential.h"

using namespace std;

int main(void) {
    
    hipSetDevice(0);

    int device;
    hipGetDevice(&device);
    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    printf("Max Threads per Block:%d\n", props.maxThreadsPerBlock);
    printf("Max Blocks per Multiprocessor:%d\n", props.maxBlocksPerMultiProcessor);
    printf("Max Shared Memory size per Block:%d bytes\n", props.sharedMemPerBlock);

    int nodes = 512;
    int* matrix = (int*)malloc(nodes * nodes * sizeof(int*));
    for (int i = 0; i < nodes; i++) {
        matrix[i] = 999;
    }

    int* resultsV1 = (int*)malloc(nodes * nodes * sizeof(int));
    int* resultsV2 = (int*)malloc(nodes * nodes * sizeof(int));

    printf("Shared Memory size used per Block:%d bytes\n", sizeof(int) * nodes + sizeof(int) * nodes * 2 + sizeof(bool) * nodes + sizeof(bool));
    generateRandomGraph(matrix, nodes);
    printf("Random graph of %d nodes initialized\n", nodes);

    /*for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            printf("%d", matrix[i * nodes + j]);
        }
        printf("\n");
    }*/

    int* gpu_matrix;
    hipError_t hipError_t = hipMalloc(&gpu_matrix, nodes * nodes * sizeof(int));

    if (hipError_t != hipSuccess) {
        printf("Errore during matrix allocation on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Matrix allocation on GPU completed\n");

    hipError_t = hipMemcpy(gpu_matrix, matrix, nodes * nodes * sizeof(int), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
        printf("Error during matrix copy on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Matrix copy on GPU completed\n");

    int* resultsMatrix;
    int* results = (int*)malloc(nodes * nodes * sizeof(int));

    hipError_t = hipMalloc(&resultsMatrix, nodes * nodes * sizeof(int));

    if (hipError_t != hipSuccess) {
        printf("Error during results matrix allocation on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Results matrix allocation completed\n");

    int* lArray;
    hipError_t = hipMalloc(&lArray, nodes * nodes * sizeof(int));

    if (hipError_t != hipSuccess) {
        printf("Error during results lArray allocation on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("lArray allocation completed\n");

    bool* VtArray;
    hipError_t = hipMalloc(&VtArray, nodes * nodes * sizeof(bool));

    if (hipError_t != hipSuccess) {
        printf("Error during results VtArray allocation on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("VtArray allocation completed\n");

    //SEQUENTIAL PART

    // Results matrix re-initialization
    for (int i = 0; i < nodes * nodes; i++) {
        results[i] = 0;
    }

    printf("\n\nSEQUENTIAL PART\n\n");

    using clock = std::chrono::system_clock;
    using ms = std::chrono::duration<double, std::milli>;
    auto before = clock::now();
    shortestPathsSequential(matrix, nodes, results);
    ms duration = clock::now() - before;

    printf("Sequential execution time: %f milliseconds\n", duration.count());

    /*for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            printf("%d", results[i * nodes + j]);
        }
        printf("\n");
    }*/
    
    //KERNEL V1 PART

    printf("\n\nKERNEL V1 PART\n\n");

    int threadsPerBlock = 1024;
    int blocks = (nodes + threadsPerBlock - 1) / threadsPerBlock;
    before = clock::now();

    shortestPathsParallel <<< blocks, threadsPerBlock >>> (gpu_matrix, nodes, resultsMatrix, lArray, VtArray);
    hipError_t = hipGetLastError();

    if (hipError_t != hipSuccess) {
        printf("Error during kernel launch: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }

    hipError_t = hipDeviceSynchronize();
    if (hipError_t != hipSuccess) {
        printf("Kernel syncronization returned error: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }

    duration = clock::now() - before;

    hipError_t = hipMemcpy(results, resultsMatrix, nodes * nodes * sizeof(int), hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess) {
        printf("Error during results copy on Host: %s\n", hipGetErrorString(hipError_t));
    }
    printf("Results copy on Host completed\n");

    /*for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            printf("%d", results[i * nodes + j]);
        }
        printf("\n");
    }*/

    for (int i = 0; i < nodes * nodes; i++) {
        resultsV1[i] = results[i];
    }

    hipFree(lArray);
    hipFree(VtArray);

    printf("Kernel V1 execution time: %f milliseconds\n", duration.count());

    // KERNEL V2 PART
    if (nodes <= 1024) {
        printf("\n\nKERNEL V2 PART\n\n");

        // Results matrix re-initialization
        for (int i = 0; i < nodes * nodes; i++) {
            results[i] = 0;
        }

        before = clock::now();
        shortestPathsParallelV2 << <nodes, nodes, sizeof(int)* nodes + sizeof(int) * nodes * 2 + sizeof(bool) * nodes + sizeof(bool) >> > (gpu_matrix, resultsMatrix);
        hipError_t = hipGetLastError();

        if (hipError_t != hipSuccess) {
            printf("Error during kernel V2 launch: %s\n", hipGetErrorString(hipError_t));
            exit(1);
        }

        hipError_t = hipPeekAtLastError();
        if (hipError_t != hipSuccess) {
            printf("Error during kernel V2 execution: %s\n", hipGetErrorString(hipError_t));
            exit(1);
        }

        hipError_t = hipDeviceSynchronize();
        if (hipError_t != hipSuccess) {
            printf("Kernel V2 syncronization returned error: %s\n", hipGetErrorString(hipError_t));
            exit(1);
        }

        duration = clock::now() - before;

        hipError_t = hipMemcpy(results, resultsMatrix, nodes * nodes * sizeof(int), hipMemcpyDeviceToHost);
        if (hipError_t != hipSuccess) {
            printf("Error during results copy on Host: %s\n", hipGetErrorString(hipError_t));
        }
        printf("Results copy on Host completed\n");

        /*for (int i = 0; i < nodes; i++) {
            for (int j = 0; j < nodes; j++) {
                printf("%d", results[i * nodes + j]);
            }
            printf("\n");
        }*/

        for (int i = 0; i < nodes * nodes; i++) {
            resultsV2[i] = results[i];
        }

        /*for (int i = 0; i < nodes; i++) {
            for (int j = 0; j < nodes; j++) {
                printf("%d", resultsV1[i * nodes + j]);
            }
            printf("\n");
        }
        printf("\n");
        for (int i = 0; i < nodes; i++) {
            for (int j = 0; j < nodes; j++) {
                printf("%d", resultsV2[i * nodes + j]);
            }
            printf("\n");
        }
        printf("\n");*/
        printf("Kernel V2 execution time: %f milliseconds\n", duration.count());
    }

    hipFree(resultsMatrix);
    hipFree(gpu_matrix);

    free(results);
    free(matrix);
    return 0;
}