#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <chrono>
#include "data.h"
#include "kernel.h"
#include "sequential.h"

using namespace std;

int main(void) {

    //Kernel part
    hipSetDevice(0);

    int nodes = 10;
    int* matrix = (int*)malloc(nodes * nodes * sizeof(int*));
    for (int i = 0; i < nodes; i++) {
        matrix[i] = 999999999;
    }

    printf("Generating random graph\n");
    generateRandomGraph(matrix, nodes);
    printf("Initial matrix loaded\n");

    for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            printf("%d ", matrix[i * nodes + j]);
        }
        printf("\n");
    }

    /*int index = 0;
    for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            matrix[index] = adj_matrix[i][j];
            index++;
        }
    }*/

    int* gpu_matrix;
    hipError_t hipError_t = hipMalloc(&gpu_matrix, nodes * nodes * sizeof(int));

    if (hipError_t != hipSuccess) {
        printf("Errore during matrix allocation on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Matrix allocation on GPU completed\n");

    hipError_t = hipMemcpy(gpu_matrix, matrix, nodes * nodes * sizeof(int), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
        printf("Error during matrix copy on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Matrix copy on GPU completed\n");

    int* resultsMatrix;
    int* results = (int*)malloc(nodes * nodes * sizeof(int));
    hipError_t = hipMalloc(&resultsMatrix, nodes * nodes * sizeof(int));

    if (hipError_t != hipSuccess) {
        printf("Error during results matrix allocation on GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Results matrix allocation completed\n");

    using clock = std::chrono::system_clock;
    using ms = std::chrono::duration<double, std::milli>;
    auto before = clock::now();

    shortestPathsParallel << <1, nodes >> > (gpu_matrix, nodes, resultsMatrix);
    hipError_t = hipGetLastError();

    if (hipError_t != hipSuccess) {
        printf("Error during kernel launch: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }

    hipError_t = hipDeviceSynchronize();
    if (hipError_t != hipSuccess) {
        printf("Kernel syncronization returned error: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }

    ms duration = clock::now() - before;

    hipError_t = hipMemcpy(results, resultsMatrix, nodes * nodes * sizeof(int), hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess) {
        printf("Error during results copy on Host: %s\n", hipGetErrorString(hipError_t));
    }
    printf("Results copy on Host completed\n");

    for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            printf("%d ", results[i * nodes + j]);
        }
        printf("\n");
    }

    hipFree(resultsMatrix);
    printf("Kernel execution time: %f milliseconds\n", duration.count());

    //Sequential part

    // Results matrix re-initialization
    for (int i = 0; i < nodes * nodes; i++) {
        results[i] = 0;
    }

    before = clock::now();
    shortestPathsSequential(matrix, nodes, results);
    duration = clock::now() - before;

    /*printf("Sequential algorithm completed\n");
    for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            printf("%d ", results[i * nodes + j]);
        }
        printf("\n");
    }*/

    printf("Sequential execution time: %f milliseconds\n", duration.count());
    free(results);
    free(matrix);
    return 0;
}