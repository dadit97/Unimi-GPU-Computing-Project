#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#define MAXWEIGHT 5

using namespace std;



int adj_matrix[10][10] = {
    {0, 1, 5, 4, 3, 2, 4, 2, 3, -1},
    {1, 0, 5, -1, 5, 3, -1, 2, -1, 5},
    {5, 5, 0, 1, -1, 3, -1, -1, -1, 4},
    {4, -1, 1, 0, -1, -1, 5, -1, -1, 3},
    {3, 5,-1,-1 ,0 ,-1 ,-1 ,-1 ,5 ,2},
    {2 ,3 ,3 ,-1 ,-1 ,0 ,-1 ,-1 ,-1 ,5},
    {4 ,-1 ,-1 ,5 ,-1 ,-1 ,0 ,-1 ,2 ,1},
    {2 ,2 ,-1 ,-1 ,-1 ,-1 ,-1 ,0 ,5 ,5},
    {3 ,-1 ,-1 ,-1 ,5 ,-1 ,2 ,5 ,0 ,-1},
    {-1 ,5 ,4 ,3 ,2 ,5 ,1 ,5 ,-1 ,0}
};

__global__ void printThreadMatrixRow(int* matrix, int dimension) {
    int tID = threadIdx.x;
    for (int i = 0; i < dimension; i++) {
        printf("Thread %d, valore %d,%d: %d\n", tID, tID, i,  matrix[tID * dimension + i]);
    }
}

__device__ bool areAllTrue(bool* vector, int dimension) {
    for (int i = 0; i < dimension; i++) {
        if (vector[i] == false) return false;
    }
    return true;
}

__device__ void initializeBoolVector(bool* vector, int dimension) {
    for (int i = 0; i < dimension; i++) {
        vector[i] = false;
    }
}

__global__ void shortestPath(int* matrix, int dimension) {
    // Each Thread computes the problem for the node with its tID index
    int tID = threadIdx.x;
    int sourceNodeIndex = tID;

    // Boolean vector simulating the Vt set initialization
    bool* Vt = (bool*)malloc((dimension) * sizeof(bool));
    initializeBoolVector(Vt, dimension);
    Vt[sourceNodeIndex] = true;

    // l vector initialization
    int* l = (int*)malloc(dimension * sizeof(int));
    
    // Getting direct connections with source node
    for (int i = 0; i < dimension; i++) {
        l[i] = matrix[sourceNodeIndex * dimension + i];
    }

    /*if (tID == 1) {
        printf("Initial l: ");
        for (int i = 0; i < dimension; i++) {
            printf("%d ", l[i]);
        }
        printf("\n");
    }*/

    
    // while V != Vt
    //while (!areAllTrue(Vt, dimension)) {
    for (int x = 0; x < 10; x++) {

        int closestWeigth = 50;
        int closestIndex = sourceNodeIndex;
        // Find the next vertex closest to source node
        for (int i = 0; i < dimension; i++) {
            if (Vt[i] == true) continue;
            if (l[i] == -1) continue;
            if (l[i] < closestWeigth) {
                closestWeigth = l[i];
                closestIndex = i;
                
            }
        }
        
        // Add closest vertex to Vt
        Vt[closestIndex] = true;

        if (tID == 0) {
            printf("Thread %d, closestIndex:%d,  closestWeigth:%d\n", tID, closestIndex, closestWeigth);
            printf("l: ");
            for (int i = 0; i < dimension; i++) {
                printf("%d ", l[i]);
            }
            printf("\n");
            printf("Vt: ");
            for (int i = 0; i < dimension; i++) {
                printf("%d ", Vt[i]);
            }
            printf("\n");
        }
        
        // Recompute l
        for (int i = 0; i < dimension; i++) {
            if (Vt[i] == true) continue;
            int uvWeight = matrix[closestIndex * dimension + i];
            if (uvWeight == -1) continue;
            l[i] = min(l[i], l[closestIndex] + uvWeight);
        }
    }
    /*for (int i = 0; i < dimension; i++) {
        if (tID == true) continue;
        printf("Thread %d, l index: %d, weight: %d \n", tID, i, l[i]);
    }*/
    free(Vt);
    free(l);
}

int main(void) {
    hipSetDevice(0);

    int nodes = 10;

    int* matrix = (int*)malloc(nodes * nodes * sizeof(int*));
    int index = 0;
    for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            matrix[index] = adj_matrix[i][j];
            index++;
        }
    }

    /* Debug array initialization
    bool* debugArray = (bool*)malloc(nodes * sizeof(bool*));
    bool* debugArrayDevice;
    hipMalloc(&debugArrayDevice, nodes * sizeof(bool));
    index = 0;
    for (int i = 0; i < nodes; i++) {
        debugArray[index] = false;
        index++;
    }
    hipMemcpy(debugArrayDevice, debugArray, nodes * sizeof(bool), hipMemcpyHostToDevice);*/

    int* gpu_matrix;
    hipError_t hipError_t = hipMalloc(&gpu_matrix, nodes * nodes * sizeof(int));

    if (hipError_t != hipSuccess) {
        printf("Errore durante l'allocazione della memoria sulla GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Allocazione della memoria sulla GPU completata\n");

    hipError_t = hipMemcpy(gpu_matrix, matrix, nodes * nodes * sizeof(int), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
        printf("Errore durante la copia della matrice sulla GPU: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
    printf("Copia della matrice sulla GPU completata\n");
    free(matrix);
        
    shortestPath<<<1, nodes >>>(gpu_matrix, nodes);

    hipDeviceSynchronize();
    return 0;
}