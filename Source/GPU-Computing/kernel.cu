#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <random>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

#ifdef __HIPCC__
#define __syncthreads() __syncthreads()
#else
#define __syncthreads()
#endif

__device__ bool areAllTrue(bool* vector, int dimension) {
    for (int i = 0; i < dimension; i++) {
        if (vector[i] == false) return false;
    }
    return true;
}

__device__ void initializeBoolVector(bool* vector, int dimension) {
    for (int i = 0; i < dimension; i++) {
        vector[i] = false;
    }
}

// FIXME
__device__ int minIndex(int* vector, int min, int indexA, int indexB) {
    return vector[indexA] == min ? indexA : indexB;
}

__device__ int minWithoutZero(int a, int b) {
    if (a == 0) return b;
    if (b == 0) return a;
    return min(a, b);
}

__global__ void shortestPathsParallel(int* matrix, int dimension, int* results) {
    // Each Thread computes the problem for the node with its tID index
    int tID = threadIdx.x;

    // Boolean vector simulating the Vt set initialization
    bool* Vt = (bool*)malloc((dimension) * sizeof(bool));
    initializeBoolVector(Vt, dimension);
    Vt[tID] = true;

    // l vector initialization
    int* l = (int*)malloc(dimension * sizeof(int));
    
    // Getting direct connections with source node
    for (int i = 0; i < dimension; i++) {
        l[i] = matrix[tID * dimension + i];
    }
    
    // while V != Vt
    while (!areAllTrue(Vt, dimension)) {

        int closestWeigth = 999999999;
        int closestIndex = tID;

        // Find the next vertex closest to source node
        for (int i = 0; i < dimension; i++) {
            if (Vt[i] == true) continue;
            if (l[i] < closestWeigth) {
                closestWeigth = l[i];
                closestIndex = i;
                
            }
        }
        
        // Add closest vertex to Vt
        Vt[closestIndex] = true;
        
        // Recompute l
        for (int i = 0; i < dimension; i++) {
            if (Vt[i] == true) continue;
            int uvWeight = matrix[closestIndex * dimension + i];
            l[i] = min(l[i], l[closestIndex] + uvWeight);
        }
    }

    for (int i = 0; i < dimension; i++) {
        results[tID * dimension + i] = l[i];
    }

    free(Vt);
    free(l);
}

__global__ void shortestPathsParallelV2(int* matrix, int dimension, int* results) {
    // Each Block computes the problem for the node with its blockID index
    // Max threads per block = 1024
    // Numero di thread per blocco = min(1024/numero nodi, nodi di una riga)
    int tID = threadIdx.x;
    int bID = blockIdx.x;
    int bDim = blockDim.x;

    // Shared memory initialization
    extern __shared__ int s[];
    int* sharedData = s;

    // l vector initialization
    int* l = (int*)&sharedData[0];

    // minimum vector initialization, first half are values second half are indexes
    int* minimum = (int*)&l[bDim];

    // Boolean vector simulating the Vt set initialization
    bool* Vt = (bool*)&minimum[bDim * 2];

    bool* stopCycle = (bool*)&Vt[bDim];
    
    Vt[tID] = false;

    if (tID == 0) {
        Vt[bID] = true;
    }
    __syncthreads();
    
    // Getting direct connections with source node
    l[tID] = matrix[bID * dimension + tID];

    __syncthreads();

    //FASE 1 : ricerca lineare del nodo pi� vicino localmente
    // IN QUESTO CASO HO UN THREAD PER NODO, QUINDI IL LOCALE E' GIA' NOTO

    //FASE 2 : Riduzione per trovare il pi� vicino globale

    while (true) {

        if (tID == 0) {
            stopCycle[0] = areAllTrue(Vt, dimension);
        }
        __syncthreads();

        if (stopCycle[0]) {
            break;
        }
        __syncthreads();

        // Restoring min shared vector from l vector
        minimum[tID] = Vt[tID] ? 99999999 : l[tID];
        minimum[tID + bDim] = tID;

        /*if (bID == 0) {
            if (tID == 0) {
                printf("\n\n\n");
            }
            printf("TID:%d,value:%d ",tID, minimum[tID]);
        }
        __syncthreads();*/

        __syncthreads();

        /*if (bID == 0 && tID == 0) {
            printf("%d - %d | %d ||", minimum[tID], minimum[blockDim.x + tID], Vt[tID]);
            printf("\n");
        }

        __syncthreads();*/

        // in-place reduction
        for (int stride = 1; stride < bDim; stride *= 2) {
            /*if (bID == 0) {
                if (tID == 0) printf("\n");
                printf("%d ", minimum[tID]);
            }
            __syncthreads();*/
            // convert tid into local array index
            int index = 2 * stride * tID;
            if (index < bDim) {

                int localMinBefore = minimum[index];
                int localMinIndexBefore = minimum[index + bDim];
                int localMin = min(minimum[index], minimum[index + stride]);
                int localMinIndex = minIndex(minimum, localMin, index, index + stride);
                minimum[index + bDim] = localMinIndex;
                minimum[index] = localMin;

                /*if (bID == 0) {
                    if (tID == 0) printf("\n\n");
                    __syncthreads();
                    printf("tID %d: %d,%d - %d,%d ", tID, localMinBefore, localMinIndexBefore, localMin, localMinIndex);
                }
                __syncthreads();*/
            }
            // synchronize within threadblock
            __syncthreads();
        }

        if (bID == 0) {
            if (tID == 0) printf("\n\n");
            __syncthreads();
            printf("%d - %d | %d ||", minimum[tID], minimum[blockDim.x + tID], Vt[tID]);
            printf("\n");
        }

        __syncthreads();

        // Add closest vertex to Vt
        if (tID == 0) {
            Vt[minimum[bDim]] = true;
            //if(bID == 0)printf("%d-%d\n", minimum[bDim], minimum[0]);
        }
        __syncthreads();

        // Recompute l
        /*if (bID == 0) {
            if (tID == 0) printf("\n\n\n");
            printf("%d,%d ", Vt[tID], l[tID]);
        }
        __syncthreads();*/
        if (!Vt[tID]) {
            /*if (bID == 0) {
                if (tID == 0) printf("\n\n\n");
                printf("%d,%d ", Vt[tID], l[tID]);
            }
            __syncthreads();*/
            int uvWeight = matrix[minimum[bDim] * dimension + tID];
            l[tID] = min(l[tID], l[minimum[bDim]] + uvWeight);
            __syncthreads();
        }

        //if (bID == 0 && tID == 0) printf("l[tID]: %d, vT[tID]: %d, closestIndex: %d\n", l[tID], Vt[tID], minimum[blockDim.x]);

        __syncthreads();
    }

    results[bID * dimension + tID] = l[tID];

    free(Vt);
    free(l);
    free(minimum);

    /* PARTE VECCHIA
    // while V != Vt
    while (!areAllTrue(Vt, dimension)) {

        int closestWeigth = 999999999;
        int closestIndex = tID;

        

        // Find the next vertex closest to source node
        if (Vt[tID] != true) {

            // in-place reduction
            for (int stride = 1; stride < blockDim.x; stride *= 2) {
                // convert tid into local array index
                int index = 2 * stride * tID;
                if (index < blockDim.x)
                    min[index] = atomicMin(min[index], min[index + stride]);
                // synchronize within threadblock
                __syncthreads();
            }

            if (l[tID] < closestWeigth) {
                closestWeigth = l[tID];
                closestIndex = tID;
            }
        }
        // Add closest vertex to Vt
        Vt[closestIndex] = true;
        __syncthreads();

        // Recompute l
        if (Vt[tID] != true) {
            int uvWeight = matrix[closestIndex * dimension + tID];
            l[tID] = min(l[tID], l[closestIndex] + uvWeight);
        }
        __syncthreads();
    } */
}